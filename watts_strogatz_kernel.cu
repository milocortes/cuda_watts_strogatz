#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 9000000
#define M 3
#define P 0.1
#define BLOCK_SIZE 8

using namespace std;

double serialTimer = 0.0;
float parallelTimer = 0.0;

// Definimos el arreglo en 1D que contendrá los valores de la matriz de adyacencia
int *h_value;
int *d_value;
// Definimos el arreglo en 1D que contendrá los indices de las columnas
int *h_colidx;
int *d_colidx;
// Definimos el arreglo en 1D que contendrá los indices de las filas
int *h_rowidx;
int *d_rowidx;

// Definimos los arreglos en 1D de la ejecución secuencial
int *secuencial_value;
int *secuencial_colidx;
int *secuencial_rowidx;

// Definimos los métodos
void crea_anillo_cpu();
void imprime_coo_cpu() ;
void imprime_coo_gpu() ;
double get_random();
bool in_edges_cpu(int node, int edge);
void watts_strogatz_cpu();
void evalua_desconexion_cpu(int node,int edge_original,int edge);
void reconecta_cpu(int node,int edge_original,int edge);
void evalua_desconexion_gpu(int node,int edge_original,int edge);
void reconecta_gpu(int node,int edge_original,int edge);
bool in_edges_gpu(int node, int edge);
void gpu_watts_strogatz();
// Kernel gpu_crea_anillo
__global__ void gpu_crea_anillo(int *value,int *rowidx, int *colidx){
  int rownum = blockIdx.x * blockDim.x + threadIdx.x;
  int colnum = blockIdx.y * blockDim.y + threadIdx.y;

  if (rownum < (N*M) && colnum > 0 && colnum <= M) {
    value[(rownum*M) + (colnum-1)] = 1 ;
    rowidx[(rownum*M) + (colnum-1)] = rownum;
    colidx[(rownum*M) +(colnum-1)] = ( rownum + colnum) % N;
  }
}

__global__ void gpu_compute_watts_strogatz(int *value,int *rowidx, int *colidx){

  int rownum = blockIdx.x * blockDim.x + threadIdx.x;
  int colnum = blockIdx.y * blockDim.y + threadIdx.y;
  /* CUDA's random number library uses hiprandState_t to keep track of the seed value
   we will store a random state for every thread  */
   hiprandState_t state;

  /* we have to initialize the state */
  hiprand_init(rownum * colnum, /* the seed controls the sequence of random values that are produced */
            blockIdx.x, /* the sequence number is only important with multiple cores */
            0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
            &state);
  if (rownum < (N*M) && colnum > 0 && colnum <= M) {
    int l = (rownum+colnum) % N;
    float aleatorio = hiprand_uniform(&state);
    printf("%.6f\n",aleatorio);
    if(aleatorio<P){
      int edge_vecino = hiprand_uniform(&state) * N;
      int *edges;
      edges = (int*)malloc(M* sizeof(int));
      int *edges_pos;
      edges_pos = (int*)malloc(M* sizeof(int));
       for(int i=0; i <M; i++){
          edges[i]=colidx[(rownum*M)+i];
          edges_pos[i]=(rownum*M)+i;
       }

       bool flag=false;
       int index = 0;

       for(int i=0; i<M; i++){
          if(edges[i]==edge_vecino){
           flag = true;
           index = edges_pos[i];
          }
       }

       if (flag !=true || edge_vecino!=rownum) {
         printf("Reconectamos la edge (%d,%d) con el nodo %d\n",rownum,l,edge_vecino);
         colidx[(rownum*M)+index]=edge_vecino;
       }

    }
  }
}

int main(int argc, char const *argv[]) {

  // Reservamos memoria para los arreglos de la ejecución secuencial
  secuencial_value = (int*)malloc( N*M* sizeof(int));
  secuencial_colidx = (int*)malloc( N*M* sizeof(int));
  secuencial_rowidx = (int*)malloc( N*M* sizeof(int));

  // Reservamos memoria para los arreglos del host
  h_value = (int*)malloc( N*M* sizeof(int));
  h_colidx = (int*)malloc( N*M* sizeof(int));
  h_rowidx = (int*)malloc( N*M* sizeof(int));

  // Ejecución Serial
  clock_t start = clock();
  crea_anillo_cpu();
  watts_strogatz_cpu();
  clock_t end = clock();
  serialTimer = double (end-start) / double(CLOCKS_PER_SEC);
  cout << "Tiempo serial: " << serialTimer << endl;
  //imprime_coo_cpu();
  // Ejecución Paralela
  std::cout << "#####################################" << '\n';
  gpu_watts_strogatz();
  //imprime_coo_gpu();
  cout << "Serial: " << serialTimer << " Parallel: " << parallelTimer / 1000 <<endl;
  cout << "Speed-up: " << serialTimer / (parallelTimer /1000)<< "X"<<endl;

  return 0;
}

void crea_anillo_cpu() {
  for (int i = 0; i < N; i++) {
    for (int j = 1; j <= M; j++) {
      secuencial_value[(i*M) + (j-1)] = 1 ;
      secuencial_rowidx[(i*M) + (j-1)] = i;
      secuencial_colidx[(i*M) +(j-1)] = ( i + j) % N;
    }
  }
}

void watts_strogatz_cpu(){
  for (int i = 0; i < N; i++) {
    for (int j = 1; j <= M; j++) {
      int l = (i+j) % N;
      if(get_random()<P){
        //std::cout<<"Entramos a evaluar"<<'\n';
        evalua_desconexion_cpu(i,l,(get_random()*N));
      }
    }
  }
}

void imprime_coo_cpu() {
  for (int i = 0; i < (N*M); i++) {
    std::cout << "Rowidx " << secuencial_rowidx[i]<< " Colidx "<< secuencial_colidx[i]<<" Valor " << secuencial_value[i]<<'\n';
  }
}

double get_random() { return ((double)rand() / (double)RAND_MAX); }

// Funciones CPU
bool in_edges_cpu(int node,int edge){
  int *edges;
  edges = (int*)malloc(M* sizeof(int));

  for(int i=0; i <M; i++){
     edges[i]=secuencial_colidx[(node*M)+i];
  }

  bool flag=false;

  for(int i=0; i<M; i++){
     if(edges[i]==edge){
      flag = true;
     }
  }

  return flag;

}

void evalua_desconexion_cpu(int node,int edge_original,int edge){
   if((in_edges_cpu(node,edge)) || (node ==edge)){
      //std::cout<<"Volvemos a evaluar"<< '\n';
      evalua_desconexion_cpu(node,edge_original,(get_random()*N));
   }else{
      //std::cout<<"Reconectamos"<< '\n';
      //std::cout<<"La arista inicial del nodo "<< node<<" es ("<< node<<"," <<edge_original<<"). Reconectamos con el nodo "<< edge<<'\n';
      reconecta_cpu(node,edge_original,edge);
   }
}
void reconecta_cpu(int node,int edge_original,int edge){

  for(int i=0; i <M; i++){
     if(secuencial_colidx[(node*M)+i]==edge_original){
         secuencial_colidx[(node*M)+i]=edge;
     }
  }

}
// Funciones GPU
void imprime_coo_gpu() {
  for (int i = 0; i < (N*M); i++) {
    std::cout << "Rowidx " << h_rowidx[i]<< " Colidx "<< h_colidx[i]<<" Valor " << h_value[i]<<'\n';
  }
}

/*
  Ejecución paralela
*/

void gpu_watts_strogatz() {
  // Reservar memoria en device
  hipMalloc((void **)&d_value, N*M * sizeof(int));
  hipMalloc((void **)&d_colidx, N*M * sizeof(int));
  hipMalloc((void **)&d_rowidx, N*M*sizeof(double));

  dim3 dimGrid((N - 1) / BLOCK_SIZE + 1, (N - 1) / BLOCK_SIZE + 1, 1);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  // define timers
  hipEvent_t start, stop;

  // events to take time
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start,0);

  gpu_crea_anillo<<<dimGrid, dimBlock>>>(d_value,d_rowidx,d_colidx);
  gpu_compute_watts_strogatz<<<dimGrid, dimBlock>>>(d_value,d_rowidx,d_colidx);
  hipMemcpy(h_value, d_value, N*M * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_rowidx, d_rowidx, N*M * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_colidx, d_colidx, N*M * sizeof(int), hipMemcpyDeviceToHost);

  hipEventRecord(stop,0);

  hipEventSynchronize(stop);

  hipEventElapsedTime(&parallelTimer, start, stop);

  cout<< "Elapsed parallel timer: " << parallelTimer << " ms, " << parallelTimer / 1000 << " secs" <<endl;

  // Copy data from device to host

}
